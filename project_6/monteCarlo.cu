#include "hip/hip_runtime.h"
// Array multiplication: C = A * B:

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block
#endif

#ifndef NUMTRIALS
#define NUMTRIALS		16384		// to make the timing more accurate
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

// ranges for the random numbers:
const float XCMIN =	 0.0;
const float XCMAX =	 2.0;
const float YCMIN =	 0.0;
const float YCMAX =	 2.0;
const float RMIN  =	 0.5;
const float RMAX  =	 2.0;

// function prototypes:
float		Ranf( float, float );
int		    Ranf( int, int );
void		TimeOfDaySeed( );

// array multiplication (CUDA Kernel) on the device: C = A * B

__global__  void ArrayMul( float *XC, float *YC, float *RS, float *C )
{
	__shared__ int hits[BLOCKSIZE];

	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	// randomize the location and radius of the circle:
	float xc = XC[gid];
	float yc = YC[gid];
	float r  = RS[gid];

	// solve for the intersection using the quadratic formula:
	float a = 2.;
	float b = -2.*( xc + yc );
	float c = xc*xc + yc*yc - r*r;
	float d = b*b - 4.*a*c;

	// CASE A: Circle is not completely missed
	if(d >= 0) {

		// hits the circle:
		// get the first intersection:
		d = sqrt( d );
		float t1 = (-b + d ) / ( 2.*a );	// time to intersect the circle
		float t2 = (-b - d ) / ( 2.*a );	// time to intersect the circle
		float tmin = t1 < t2 ? t1 : t2;		// only care about the first intersection

		// CASE B: Circle does not completely engulf the laser pointer
		if(tmin >= 0) {

			// where does it intersect the circle?
			float xcir = tmin;
			float ycir = tmin;

			// get the unitized normal vector at the point of intersection:
			float nx = xcir - xc;
			float ny = ycir - yc;
			float n = sqrt( nx*nx + ny*ny );
			nx /= n;	// unit vector
			ny /= n;	// unit vector

			// get the unitized incoming vector:
			float inx = xcir - 0.;
			float iny = ycir - 0.;
			float in = sqrt( inx*inx + iny*iny );
			inx /= in;	// unit vector
			iny /= in;	// unit vector

			// get the outgoing (bounced) vector:
			float dot = inx*nx + iny*ny;
			float outy = iny - 2.*ny*dot;	// angle of reflection = angle of incidence`

			// find out if it hits the infinite plate:
			float t = ( 0. - ycir ) / outy;

			// CASE C (false): Reflected beam went up instead of down
			// CASE D (true): Beam hit the infinite plate
			if(t >= 0) {
				hits[tnum] = 1.;
			} else {
				hits[tnum] = 0.;
			}
		} else {
			hits[tnum] = 0.;
		}
	} else {
		hits[tnum] = 0.;
	}

	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			hits[tnum] += hits[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		C[wgNum] = hits[0];
}


// main program:

int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);

	TimeOfDaySeed( );		// seed the random number generator

	FILE* outfile = fopen("results.txt", "a");
    if(!outfile) {
        fprintf(stderr, "Error opening results.txt\n");
        return -1;
    }

	// allocate host memory:

	float * hXC = new float [ NUMTRIALS ];
	float * hYC = new float [ NUMTRIALS ];
	float * hRS = new float [ NUMTRIALS ];
	float * hC  = new float [ NUMTRIALS / BLOCKSIZE ];

	// fill the random-value arrays:
    for( int n = 0; n < NUMTRIALS; n++ )
    {       
        hXC[n] = Ranf( XCMIN, XCMAX );
        hYC[n] = Ranf( YCMIN, YCMAX );
		hRS[n] = Ranf(  RMIN,  RMAX );
	}
	
	for( int n = 0; n < NUMTRIALS / BLOCKSIZE; n++) {
		hC[n]  = 0.;
	}

	// allocate device memory:

	float *dXC, *dYC, *dRS, *dC;

	dim3 dimsXC( NUMTRIALS, 1, 1 );
	dim3 dimsYC( NUMTRIALS, 1, 1 );
	dim3 dimsRS( NUMTRIALS, 1, 1 );
	dim3 dimsC( NUMTRIALS/BLOCKSIZE, 1, 1 );

	//__shared__ float prods[SIZE/BLOCKSIZE];


	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dXC), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dYC), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dRS), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dC), (NUMTRIALS/BLOCKSIZE)*sizeof(float) );
		checkCudaErrors( status );


	// copy host memory to the device:

	status = hipMemcpy( dXC, hXC, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dYC, hYC, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dRS, hRS, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( NUMTRIALS / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:

	ArrayMul<<< grid, threads >>>( dXC, dYC, dRS, dC );

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double trialsPerSecond = (float)NUMTRIALS / secondsTotal;
	double megaTrialsPerSecond = trialsPerSecond / 1000000.;
	fprintf( stderr, "Number of trials = %10d, MegaTrials/Second = %10.2lf\n", NUMTRIALS, megaTrialsPerSecond );
	fprintf(outfile, "%f\t", megaTrialsPerSecond);

	// copy result from the device to the host:

	status = hipMemcpy( hC, dC, (NUMTRIALS/BLOCKSIZE)*sizeof(float), hipMemcpyDeviceToHost );
		checkCudaErrors( status );

	// check the sum :

	int numHits = 0.;
	for(int i = 0; i < NUMTRIALS/BLOCKSIZE; i++ )
	{
		//printf("Hits at %d:\t%d\n", i, (int)hC[i]);
		numHits += (int)hC[i];
	}
	printf("Numhits: %d\n", numHits);
	fprintf( stderr, "\nprobability = %10.2lf \n", (double)numHits / (double)NUMTRIALS );

	// clean up memory:
	delete [ ] hXC;
	delete [ ] hYC;
	delete [ ] hRS;
	delete [ ] hC;

	status = hipFree( dXC );
		checkCudaErrors( status );
	status = hipFree( dYC );
		checkCudaErrors( status );
	status = hipFree( dRS );
		checkCudaErrors( status );
	status = hipFree( dC );
		checkCudaErrors( status );


	return 0;
}

float
Ranf( float low, float high )
{
    float r = (float) rand();               // 0 - RAND_MAX
    float t = r  /  (float) RAND_MAX;       // 0. - 1.

    return   low  +  t * ( high - low );
}

int
Ranf( int ilow, int ihigh )
{
    float low = (float)ilow;
    float high = ceil( (float)ihigh );

    return (int) Ranf(low,high);
}

void
TimeOfDaySeed( )
{
    struct tm y2k = { 0 };
    y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
    y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

    time_t  timer;
    time( &timer );
    double seconds = difftime( timer, mktime(&y2k) );
    unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
    srand( seed );
}
